#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 

__global__ void generate( hiprandState* globalState, float* array ) 
{
    int ind = threadIdx.x;
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState; 
    array[ind] = RANDOM;
}

int main( int argc, char** argv) 
{
    float   *host;
    float   *dev;  

    int i;
    int N = 3;

    dim3 tpb(N,1,1);
    hiprandState* devStates;
    hipMalloc ( &devStates, N*sizeof( hiprandState ) );
    
    // setup seeds
    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );

    host = (float*)malloc(sizeof(float) * N);
    hipMalloc( (void**)&dev, sizeof(float) * N);

    for(i=0; i<N; i++)
    {
      host[i] = 0;
    }

    // generate random numbers
    generate <<< 1, tpb >>> ( devStates, dev );

    hipMemcpy( host, dev, N*sizeof(float), hipMemcpyDeviceToHost );

    for(i=0; i<N; i++)
    {
      printf("Random %d: %f\n", i, host[i]);
    }

    return 0;
}
